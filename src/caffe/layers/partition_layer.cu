#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void block_copy_kernel(const int N, const int sH, const int sW, 
      const int dH, const int dW, const int sup, const int sleft, 
      const int dup, const int dleft, const int tH, const int tW, 
      const Dtype* src, Dtype* dst) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (x < N && y < tH && z < tW) {
    dst[x*dH*dW + (y+dup)*dW + z+dleft] = src[x*sH*sW + (y+sup)*sW + z+sleft];
  }
}

template <typename Dtype>
__global__ void block_add_kernel(const int N, const int sH, const int sW, 
      const int dH, const int dW, const int sup, const int sleft, 
      const int dup, const int dleft, const int tH, const int tW, 
      const Dtype* src, Dtype* dst) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (x < N && y < tH && z < tW) {
    dst[x*dH*dW + (y+dup)*dW + z+dleft] += src[x*sH*sW + (y+sup)*sW + z+sleft];
  }
}

template <typename Dtype>
void PartitionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data(); 
  const int N = bottom[0]->num();
  const int C = bottom[0]->channels();
  const int H = bottom[0]->height();
  const int W = bottom[0]->width();
  int i, j, k, l;
  int tH, tW;
  Dtype* top_data;
  for (i = 0; i < left_.size(); ++i){
    top_data = top[i]->mutable_gpu_data();
    tH = top[i]->height();
    tW = top[i]->width();
    dim3 dimBlock(CAFFE_CUDA_NUM_THREADS/256, 16, 16);
    dim3 dimGrid((N*C + dimBlock.x - 1)/dimBlock.x, 
                 (tH + dimBlock.y - 1)/dimBlock.y,
                 (tW + dimBlock.z - 1)/dimBlock.z);
    block_copy_kernel<Dtype><<<dimGrid, dimBlock>>>(N*C, H, W, tH, tW, up_[i], 
                  left_[i], 0, 0, tH, tW, bottom_data, top_data);
/*
    for (j = 0; j < N; ++j){
      for (k = 0; k < C; ++k){
        for (l = up_[i]; l < down_[i]; ++l){
          caffe_copy(right_[i] - left_[i], 
                     bottom_data + j*C*H*W + k*H*W + l*W + left_[i],
                     top_data + j*C*tH*tW + k*tH*tW + (l-up_[i])*tW);
        }
      }
    }
*/
  }
}

template <typename Dtype>
void PartitionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  caffe_set(bottom[0]->count(), Dtype(0.), bottom_diff);
  const Dtype* top_diff; 
  const int N = bottom[0]->num();
  const int C = bottom[0]->channels();
  const int H = bottom[0]->height();
  const int W = bottom[0]->width();
  int i;
  int tH, tW;
  for (i = 0; i < left_.size(); ++i){
    top_diff = top[i]->gpu_diff();
    tH = top[i]->height();
    tW = top[i]->width();
    dim3 dimBlock(CAFFE_CUDA_NUM_THREADS/256, 16, 16);
    dim3 dimGrid((N*C + dimBlock.x - 1)/dimBlock.x, 
                 (tH + dimBlock.y - 1)/dimBlock.y,
                 (tW + dimBlock.z - 1)/dimBlock.z);
    block_add_kernel<Dtype><<<dimGrid, dimBlock>>>(N*C, tH, tW, H, W, 0, 0, up_[i],
            left_[i], tH, tW, top_diff, bottom_diff);
/*
    for (j = 0; j < N; ++j){
      for (k = 0; k < C; ++k){
        for (l = up_[i]; l < down_[i]; ++l){
          caffe_gpu_add(right_[i] - left_[i], 
                     bottom_diff + j*C*H*W + k*H*W + l*W + left_[i],
                     top_diff + j*C*tH*tW + k*tH*tW + (l-up_[i])*tW,
                     bottom_diff + j*C*H*W + k*H*W + l*W + left_[i]);
        }
      }
    }
*/
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PartitionLayer);

} // namespace caffe
