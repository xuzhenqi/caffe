#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <hipblas.h>

#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
	__global__ void APLForwardSum(const int n, int s, const Dtype* in, Dtype* out, const Dtype* neuron_weight, const Dtype* neuron_offset, Dtype* maxs_data, int sums_, int K_) {
		CUDA_KERNEL_LOOP(index, n) {
			int exPos = ((int) index / K_) * K_;
			int exPosSums = exPos*sums_;
			int k = index % K_;
			int sumPos = k*sums_;

			if (s == 0) {
				out[index] = in[index] > 0 ? in[index] : 0;
			}
			maxs_data[exPosSums + sumPos + s] = max(-in[index] + neuron_offset[sumPos + s], Dtype(0));
			out[index] += neuron_weight[sumPos + s]*maxs_data[exPosSums + sumPos + s];
		}
	}

template <typename Dtype>
	__global__ void APLForwardSumHardcode(const int n, const Dtype* in, Dtype* out, const Dtype* neuron_weight, const Dtype* neuron_offset, Dtype* maxs_data, int sums_, int K_) {
		CUDA_KERNEL_LOOP(index, n) {
			int exPos = ((int) index / K_) * K_;
			int exPosSums = exPos*sums_;
			int k = index % K_;
			int sumPos = k*sums_;

			switch (sums_) {
				case 1 : { 
									 maxs_data[exPosSums + sumPos + 0] = max(-in[index] + neuron_offset[sumPos + 0], Dtype(0));

									 Dtype inMax = in[index] > 0 ? in[index] : 0;
									 out[index] = inMax +  neuron_weight[sumPos + 0]*maxs_data[exPosSums + sumPos + 0];
									 break;
								 }
				case 2 : {
									 maxs_data[exPosSums + sumPos + 0] = max(-in[index] + neuron_offset[sumPos + 0], Dtype(0));
									 maxs_data[exPosSums + sumPos + 1] = max(-in[index] + neuron_offset[sumPos + 1], Dtype(0));

									 Dtype inMax = in[index] > 0 ? in[index] : 0;
									 out[index] = inMax +  neuron_weight[sumPos + 0]*maxs_data[exPosSums + sumPos + 0] + neuron_weight[sumPos + 1]*maxs_data[exPosSums + sumPos + 1];
									 break;
								 }
				case 3 : {
									 maxs_data[exPosSums + sumPos + 0] = max(-in[index] + neuron_offset[sumPos + 0], Dtype(0));
									 maxs_data[exPosSums + sumPos + 1] = max(-in[index] + neuron_offset[sumPos + 1], Dtype(0));
									 maxs_data[exPosSums + sumPos + 2] = max(-in[index] + neuron_offset[sumPos + 2], Dtype(0));

									 Dtype inMax = in[index] > 0 ? in[index] : 0;
									 out[index] = inMax +  neuron_weight[sumPos + 0]*maxs_data[exPosSums + sumPos + 0] + neuron_weight[sumPos + 1]*maxs_data[exPosSums + sumPos + 1] + neuron_weight[sumPos + 2]*maxs_data[exPosSums + sumPos + 2];
									 break;
								 }
				case 4 : {
									 maxs_data[exPosSums + sumPos + 0] = max(-in[index] + neuron_offset[sumPos + 0], Dtype(0));
									 maxs_data[exPosSums + sumPos + 1] = max(-in[index] + neuron_offset[sumPos + 1], Dtype(0));
									 maxs_data[exPosSums + sumPos + 2] = max(-in[index] + neuron_offset[sumPos + 2], Dtype(0));
									 maxs_data[exPosSums + sumPos + 3] = max(-in[index] + neuron_offset[sumPos + 3], Dtype(0));

									 Dtype inMax = in[index] > 0 ? in[index] : 0;
									 out[index] = inMax +  neuron_weight[sumPos + 0]*maxs_data[exPosSums + sumPos + 0] + neuron_weight[sumPos + 1]*maxs_data[exPosSums + sumPos + 1] + neuron_weight[sumPos + 2]*maxs_data[exPosSums + sumPos + 2] + neuron_weight[sumPos + 3]*maxs_data[exPosSums + sumPos + 3];
									 break;
								 }
				case 5 : {
									 maxs_data[exPosSums + sumPos + 0] = max(-in[index] + neuron_offset[sumPos + 0], Dtype(0));
									 maxs_data[exPosSums + sumPos + 1] = max(-in[index] + neuron_offset[sumPos + 1], Dtype(0));
									 maxs_data[exPosSums + sumPos + 2] = max(-in[index] + neuron_offset[sumPos + 2], Dtype(0));
									 maxs_data[exPosSums + sumPos + 3] = max(-in[index] + neuron_offset[sumPos + 3], Dtype(0));
									 maxs_data[exPosSums + sumPos + 4] = max(-in[index] + neuron_offset[sumPos + 4], Dtype(0));

									 Dtype inMax = in[index] > 0 ? in[index] : 0;
									 out[index] = inMax +  neuron_weight[sumPos + 0]*maxs_data[exPosSums + sumPos + 0] + neuron_weight[sumPos + 1]*maxs_data[exPosSums + sumPos + 1] + neuron_weight[sumPos + 2]*maxs_data[exPosSums + sumPos + 2] + neuron_weight[sumPos + 3]*maxs_data[exPosSums + sumPos + 3] + neuron_weight[sumPos + 4]*maxs_data[exPosSums + sumPos + 4];
									 break;
								 }
			}
		}
	}

template <typename Dtype>
	void APLLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
			const vector<Blob<Dtype>*>& top) {
		//Forward_cpu(bottom,top);

		//Initialize
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();

		Dtype* maxs_data = reinterpret_cast<Dtype*>(maxs_->mutable_gpu_data());
		const int count = bottom[0]->count();

		//For in-place computation
		if (bottom[0] == top[0]) {
			caffe_copy(count, bottom_data, inPlace_memory_.mutable_gpu_data());
			bottom_data = inPlace_memory_.gpu_data();
		}

		const Dtype* neuron_weight = this->blobs_[0]->gpu_data();
		const Dtype* neuron_offset = this->blobs_[1]->gpu_data();

		for (int s=0; s<sums_; s++) {
			APLForwardSum<Dtype><<<CAFFE_GET_BLOCKS(M_*K_), CAFFE_CUDA_NUM_THREADS>>>(
					M_*K_, s, bottom_data, top_data,neuron_weight,neuron_offset,maxs_data,sums_,K_);
			CUDA_POST_KERNEL_CHECK;
		}
	}

template <typename Dtype>
	__global__ void ComputeDiffExample(int n, int e, Dtype* neuron_weight_diff, Dtype* neuron_offset_diff, const Dtype* neuron_weight, const Dtype* top_diff, const Dtype* bottom_data, const Dtype* maxs_data, int sums_, int K_) {
		CUDA_KERNEL_LOOP(index, n) {
			int k = index / sums_;
			Dtype maxGT_Zero = maxs_data[index] > 0;
			Dtype offset_diff = top_diff[k]*neuron_weight[index]*maxGT_Zero;

			if (e == 0) {
				neuron_weight_diff[index] = top_diff[k]*maxs_data[index];
				neuron_offset_diff[index] = offset_diff;
			} else {
				neuron_weight_diff[index] += top_diff[k]*maxs_data[index];
				neuron_offset_diff[index] += offset_diff;
			}
		}
	}

template <typename Dtype>
	__global__ void ComputeOffsetDiff(int n, Dtype* offset_diff, const Dtype* neuron_weight, const Dtype* top_diff, const Dtype* maxs_data, int sums_, int M_, int K_, int K_Times_Sums) {
		CUDA_KERNEL_LOOP(index, n) {
			Dtype maxGT_Zero = maxs_data[index] > 0;
			offset_diff[index] = top_diff[index / sums_]*neuron_weight[index % K_Times_Sums]*maxGT_Zero;
		}
	}

template <typename Dtype>
	__global__ void ComputeBottomDiffSum(int n, int s, Dtype* bottom_diff, const Dtype* bottom_data, const Dtype* neuron_weight, const Dtype* maxs_data, const Dtype* top_diff, int sums_, int K_) {
		CUDA_KERNEL_LOOP(index, n) {
			int exPos = ((int) index / K_) * K_;
			int exPosSums = exPos*sums_;
			int k = index % K_;
			int sumPos = k*sums_;

			Dtype maxGT_Zero = maxs_data[exPosSums + sumPos + s] > 0;

			Dtype offset_diff = top_diff[index]*neuron_weight[sumPos + s]*maxGT_Zero;

			if (s == 0) {
				bottom_diff[index] = bottom_data[index] > 0 ? top_diff[index] : 0;
			}
			bottom_diff[index] += -offset_diff;
		}
	}

template <typename Dtype>
	__global__ void PropDownMax(int n, Dtype* bottom_diff, const Dtype* bottom_data, const Dtype* top_diff) {
		CUDA_KERNEL_LOOP(index, n) {
			bottom_diff[index] = bottom_data[index] > 0 ? top_diff[index] : 0;
		}
	}

//Mimics Matlab's bsxfun with only mult implemented for now
template <typename Dtype>
	__global__ void bsxfun(int n, const Dtype* mat1, const Dtype* vec, const int mat1dim, const int vectorDim, Dtype* resultMat) {
		CUDA_KERNEL_LOOP(index, n) {
			int vecPos = index % vectorDim;
			int mat1pos2 = index/vectorDim;
			int matLoc = vecPos*mat1dim + mat1pos2;

			resultMat[matLoc] = mat1[matLoc]*vec[vecPos];
		}
	}

template <typename Dtype>
	void APLLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
			const vector<bool>& propagate_down,
			const vector<Blob<Dtype>*>& bottom) {
		//Backward_cpu(top,propagate_down,bottom);

		//Initialize
		const Dtype* bottom_data = bottom[0]->gpu_data();
		const Dtype* top_diff = top[0]->gpu_diff();
		const int count = bottom[0]->count();
		if (top[0] == bottom[0]) {
			bottom_data = inPlace_memory_.gpu_data();
			caffe_copy(count, top_diff, inPlace_memory_.mutable_gpu_diff());
			top_diff = inPlace_memory_.gpu_diff();
		}

		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

		// For in-place computation
		Dtype* maxs_data = reinterpret_cast<Dtype*>(maxs_->mutable_gpu_data());

		//Backprop apl weights
		const Dtype* neuron_weight = this->blobs_[0]->gpu_data();

		Dtype* neuron_weight_diff = this->blobs_[0]->mutable_gpu_diff();
		Dtype* neuron_offset_diff = this->blobs_[1]->mutable_gpu_diff();

		if (save_mem_) {
			for (int e=0; e<M_; ++e) {
				ComputeDiffExample<Dtype><<<CAFFE_GET_BLOCKS(K_*sums_), CAFFE_CUDA_NUM_THREADS>>>(K_*sums_, e, neuron_weight_diff, neuron_offset_diff, neuron_weight, top_diff + e*K_, bottom_data + e*K_, maxs_data + e*K_*sums_, sums_, K_);
				CUDA_POST_KERNEL_CHECK;
			}
		} else {

			Dtype* temp_ex_neuron_sum = reinterpret_cast<Dtype*>(temp_ex_neuron_sum_->mutable_gpu_data());
			const Dtype* example_multiplier = reinterpret_cast<const Dtype*>(example_multiplier_->gpu_data());

			//Compute derivative for neuron_weight
			bsxfun<Dtype><<<CAFFE_GET_BLOCKS(M_*K_*sums_), CAFFE_CUDA_NUM_THREADS>>>(M_*K_*sums_, maxs_data, top_diff, sums_, M_*K_, temp_ex_neuron_sum);
			CUDA_POST_KERNEL_CHECK;

			caffe_gpu_gemv<Dtype>(CblasTrans, M_, K_*sums_, (Dtype)1., temp_ex_neuron_sum,
					example_multiplier, (Dtype)0., neuron_weight_diff);

			//Compute derivative for neuron_offset
			ComputeOffsetDiff<Dtype><<<CAFFE_GET_BLOCKS(M_*K_*sums_), CAFFE_CUDA_NUM_THREADS>>>(M_*K_*sums_, temp_ex_neuron_sum, neuron_weight, top_diff, maxs_data, sums_, M_, K_, K_*sums_);
			CUDA_POST_KERNEL_CHECK;

			caffe_gpu_gemv<Dtype>(CblasTrans, M_, K_*sums_, (Dtype)1., temp_ex_neuron_sum,
					example_multiplier, (Dtype)0., neuron_offset_diff);

		}

		// Compute derivative to bottom
		if (propagate_down[0]) {
			for (int s=0; s<sums_; ++s) {
				ComputeBottomDiffSum<Dtype><<<CAFFE_GET_BLOCKS(M_*K_), CAFFE_CUDA_NUM_THREADS>>>(M_*K_, s, bottom_diff, bottom_data, neuron_weight, maxs_data, top_diff, sums_, K_);
				CUDA_POST_KERNEL_CHECK;
			}
		}
	}

INSTANTIATE_LAYER_GPU_FUNCS(APLLayer);

}  // namespace caffe
